#include "hip/hip_runtime.h"
﻿#include "macro.cuh"
using namespace std;

__global__
void init(const int pages, float* r, float* r_, float* y) {
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < pages) {
		r[tid] = 1.f / pages;
		y[tid] = 0.f;
		r_[tid] = 0.f;
	}
}

__global__
void SpMV(
	const int pages,		// number of total pages
	float* r,				// pagerank value of each page
	float* y,				// temp value before r calculated
	float* r_,				// r value of last iteration
	const int* col,			// column index of each edge
	const int* rowptr,		// row pointer of each row (CSR)
	const float* value)		// value of each edge	
{
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < pages) {
		int r_begin = rowptr[tid];
		int r_end = rowptr[tid + 1];
		float acc = 0.f;
		for (int c = r_begin; c < r_end; c++) {
			float a = value[c];
			float b = r[col[c]];
			acc += a * b;
		}
		y[tid] = Q * acc;
		atomicAdd(&y[pages], y[tid]);
	}
}
__global__ 
void renew(const int pages, float* y) {
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid == 0)
		y[pages] = 0.f;
}

__global__
void param_update(const int pages, float *r, float *r_, float *y)
{
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < pages) {
		r_[tid] = r[tid];
		r[tid] = y[tid] + (1.f - y[pages]) / pages;
	}
}

void power_iter_cu(
	const int pages,		// number of total pages
	const int nonzeros,		// number of total edges
	float* r,				// pagerank value of each page
	const int* row,			// row index of each edge
	const int* col,			// column index of each edge
	const int* rowptr,		// row pointer of each row (CSR)
	const int* rowcount,	// number of edges in each row 
	const float* value)		// value of each edge		
{
	int deviceId;
	hipGetDevice(&deviceId);
	const int blocks = pages / THREADS_PER_BLOCK + 1;
	float* _r, *_r_, *_y, *_Av;
	int* _Ap, * _Aj;

	hipMalloc(&_r, sizeof(float) * pages);
	hipMalloc(&_r_, sizeof(float) * pages);
	hipMalloc(&_y, sizeof(float) * (1 + pages));
	
#ifdef prefetch
	hipMemPrefetchAsync(rowptr, sizeof(int) * (pages + 1), deviceId);
	hipMemPrefetchAsync(col, sizeof(int) * (nonzeros), deviceId);
	hipMemPrefetchAsync(value, sizeof(float) * (nonzeros), deviceId);
	init << < blocks, THREADS_PER_BLOCK >> > (pages, _r, _r_, _y);
	for (int i = 0; i < ITER; i++) {
		renew << < blocks, THREADS_PER_BLOCK >> > (pages, _y);
		SpMV << < blocks, THREADS_PER_BLOCK >> > (pages, _r, _y, _r_, col, rowptr, value);
		param_update << < blocks, THREADS_PER_BLOCK >> > (pages, _r, _r_, _y);
	};
	hipMemPrefetchAsync(r, sizeof(float) * pages, hipCpuDeviceId);
#else
	hipMalloc(&_Ap, sizeof(int) * (pages + 1));
	hipMalloc(&_Aj, sizeof(int) * (nonzeros));
	hipMalloc(&_Av, sizeof(float) * (nonzeros));
	hipMemcpy(_Ap, rowptr, sizeof(int) * (pages + 1), hipMemcpyHostToDevice);
	hipMemcpy(_Aj, col, sizeof(int) * (nonzeros), hipMemcpyHostToDevice);
	hipMemcpy(_Av, value, sizeof(float) * (nonzeros), hipMemcpyHostToDevice);

	init << < blocks, THREADS_PER_BLOCK >> > (pages, _r, _r_, _y);
	for (int i = 0; i < ITER; i++) {
		renew << < blocks, THREADS_PER_BLOCK >> > (pages, _y);
		SpMV << < blocks, THREADS_PER_BLOCK >> > (pages, _r, _y, _r_, _Aj, _Ap, _Av);
		param_update << < blocks, THREADS_PER_BLOCK >> > (pages, _r, _r_, _y);
	};
	hipMemcpy(r, _r, sizeof(float) * pages, hipMemcpyDeviceToHost);
	hipFree(_Ap);
	hipFree(_Aj);
	hipFree(_Av);
#endif
	
	hipDeviceSynchronize();
	hipFree(_r);
	hipFree(_r_);
	hipFree(_y);
	
}