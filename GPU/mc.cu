#include "hip/hip_runtime.h"
﻿#include "macro.cuh"

using namespace std;
__global__
void mc_init(const int pages, float* r, hiprandStateMRG32k3a* state, unsigned long seed) {
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < pages)
		r[tid] = 0;

	if (threadIdx.x < WARP_SIZE) {
		int rid = threadIdx.x + blockIdx.x * WARP_SIZE;
		hiprand_init(seed, rid, 0, &state[rid]);
	}
}

__global__
void rand_walk(
	const int pages,			// number of total pages
	const int nonzeros,			// number of total edges
	float* r,					// pagerank value of each page
	const int* col,				// column index of each edge
	const int* rowptr,			// row pointer of each row (CSR)
	hiprandStateMRG32k3a* state)	// state of random number generator
{
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	const int rid = (threadIdx.x % WARP_SIZE) + blockIdx.x * WARP_SIZE;

	if (tid < pages) {
		int cur = tid;
		for (int i = 0; i < ITERATIONS; i++) {
			int count = rowptr[cur + 1] - rowptr[cur];
			if (hiprand_uniform(&state[rid]) < Q) {
				cur = count == 0 ? cur : col[rowptr[cur] + (int)(hiprand_uniform(&state[rid]) * count)];
			}
			else
				cur = tid;
			atomicAdd(&r[cur], 1);
		}
	}
}

__global__
void norm(const int pages, float* r, float *sum) {
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < pages) {
		r[tid] /= ITERATIONS;
	}
}

void monte_carlo_cu(
	const int pages,		// number of total pages
	const int nonzeros,		// number of total edges
	float* r,				// pagerank value of each page
	const int* row,			// row index of each edge
	const int* col,			// column index of each edge
	const int* rowptr,		// row pointer of each row (CSR)
	const int* rowcount,	// number of edges in each row 
	const float* value)		// value of each edge
{
	int deviceId;
	hipGetDevice(&deviceId);
	float* _r;
	int* _Ap, * _Aj;
	hiprandStateMRG32k3a* state;

	float* sum = new float(0);
	const int blocks = pages / THREADS_PER_BLOCK + 1;
	hipMalloc(&state, sizeof(hiprandStateMRG32k3a) * blocks * WARP_SIZE);
	hipMalloc(&_r, sizeof(float) * pages);
	
#ifdef prefetch
	hipMemPrefetchAsync(rowptr, sizeof(int) * (pages + 1), deviceId);
	hipMemPrefetchAsync(col, sizeof(int) * (nonzeros), deviceId);
	mc_init << < blocks, THREADS_PER_BLOCK >> > (pages, _r, state, time(NULL));
	rand_walk << < blocks, THREADS_PER_BLOCK >> > (pages, nonzeros, _r, col, rowptr, state);
	norm << < blocks, THREADS_PER_BLOCK >> > (pages, _r, sum);
	hipMemPrefetchAsync(r, sizeof(float) * pages, hipCpuDeviceId);
#else
	hipMalloc(&_Ap, sizeof(int) * (pages + 1));
	hipMalloc(&_Aj, sizeof(int) * (nonzeros));
	hipMemcpy(_Ap, rowptr, sizeof(int) * (pages + 1), hipMemcpyHostToDevice);
	hipMemcpy(_Aj, col, sizeof(int) * (nonzeros), hipMemcpyHostToDevice);

	mc_init <<< blocks, THREADS_PER_BLOCK >>> (pages, _r, state, time(NULL));
	rand_walk <<< blocks, THREADS_PER_BLOCK >>> (pages, nonzeros, _r, _Aj, _Ap, state);
	norm <<< blocks, THREADS_PER_BLOCK >>> (pages, _r, sum);
	hipMemcpy(r, _r, sizeof(float) * pages, hipMemcpyDeviceToHost);
	hipFree(_Ap);
	hipFree(_Aj);
#endif
	hipDeviceSynchronize();
	
	hipFree(state);
	hipFree(_r);
	
}